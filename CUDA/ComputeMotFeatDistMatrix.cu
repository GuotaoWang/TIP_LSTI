#include "hip/hip_runtime.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include ""
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v8.0\include\hip/hip_runtime.h"
 #include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v8.0\include\"
#include <stdio.h>
#include <algorithm>


__global__ void FeatDist_kernel(double *Result,double* MiddlePoint,double *spnum)
{
   int K=(int)(*spnum);
	int j =  threadIdx.x;
	if (threadIdx.x >= K)
		return;

	int k;
	double LlocationVx,LlocationVy;
	double RlocationVx,RlocationVy;
    double LDist=0;

    LlocationVx = MiddlePoint[j],LlocationVy = MiddlePoint[K+j];
	for (k=0;k<K;k++)
	{
		RlocationVx = MiddlePoint[k],RlocationVy = MiddlePoint[K+k];
		LDist=abs(LlocationVx-RlocationVx)+abs(LlocationVy-RlocationVy);
	    Result[j * K + k] = LDist;
	}

	return;

}
extern void ComputeMotFeatDistMatrix(double *Result, double* MiddlePoint,double *spnum);
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	double *MiddlePoint;
	double *Result;
    double *spnum;
	MiddlePoint=mxGetPr(prhs[0]);
    spnum=mxGetPr(prhs[1]);
    int K=(int)(*spnum);
	plhs[0]=mxCreateDoubleMatrix( K, K,mxREAL);
	Result=mxGetPr(plhs[0]);
	ComputeMotFeatDistMatrix(Result,MiddlePoint, spnum);
}
void ComputeMotFeatDistMatrix(double *Result, double* MiddlePoint,double *spnum)
{
	double * dev_Result;
	double *dev_mid;
    double *dev_spnum;
    int K=(int)(*spnum);
	hipMalloc((void **)&dev_mid, sizeof(double)* K *2);
	hipMalloc((void **)&dev_Result, sizeof(double)*K*K);
    hipMalloc((void **)&dev_spnum, sizeof(double));

	hipMemcpy(dev_mid, MiddlePoint, sizeof(double)* K*2, hipMemcpyHostToDevice);
	hipMemcpy(dev_spnum, spnum, sizeof(double), hipMemcpyHostToDevice);

	dim3 threads(K);
	FeatDist_kernel << <1, threads >> >(dev_Result,dev_mid,dev_spnum);
    hipMemcpy(Result, dev_Result, sizeof(double)*K*K, hipMemcpyDeviceToHost);

	hipFree(dev_mid);
    hipFree(dev_Result);
    hipFree(dev_spnum);

}	




