#include "hip/hip_runtime.h"
#include "mex.h"
#include "hip/hip_runtime.h"
#include ""
#include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v8.0\include\hip/hip_runtime.h"
 #include "C:\Program Files\NVIDIA GPU Computing Toolkit\CUDA\v8.0\include\"
#include <stdio.h>
#include <algorithm>


__global__ void smooth_kernel(double *Result, double* midP,double* Ind,double *K1,double *SaA,double *SaA0,double *SaA2,double *N1,double *P1,double *P2,double *spnum)
{
	int j =  threadIdx.x;
    int K=(int)(*K1),i=(int)(*Ind), N=(int)(*N1);
	if (threadIdx.x >= K)
		return;

	int k;
	double Lcolor1,Lcolor2,Lcolor3,Lcolor4,Lcolor5,Llocation1,Llocation2;
	double Rcolor1,Rcolor2,Rcolor3,Rcolor4,Rcolor5,Rlocation1,Rlocation2;
	double Tweight=0,weight1=0;
	double MSS=0;
	double CDist=0,LDist=0;
    Llocation1 = midP[(K)*7*i+j],Llocation2 = midP[(K)*7*i+(K)*1+j],Lcolor1 = midP[(K)*7*i+(K)*2+j], Lcolor2 = midP[(K)*7*i+(K)*3+j],Lcolor3 = midP[(K)*7*i+(K)*4+j],Lcolor4 = midP[(K)*7*i+(K)*5+j],Lcolor5 = midP[(K)*7*i+(K)*6+j];
	for (k=0;k<spnum[i];k++)
	{
		Rlocation1 = midP[(K)*7*i+k],Rlocation2 = midP[(K)*7*i+(K)*1+k],Rcolor1 = midP[(K)*7*i+(K)*2+k], Rcolor2 = midP[(K)*7*i+(K)*3+k],Rcolor3 = midP[(K)*7*i+(K)*4+k],Rcolor4 = midP[(K)*7*i+(K)*5+k],Rcolor5 = midP[(K)*7*i+(K)*6+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2)+(Lcolor4-Rcolor4)*(Lcolor4-Rcolor4)+(Lcolor5-Rcolor5)*(Lcolor5-Rcolor5));
		if (LDist<(*P1))
		{
			weight1=exp(-CDist*(*P2));
			Tweight+=weight1;
			MSS=MSS+SaA[k]*weight1;
		}
	}
    if(i<N-1)
    for ( k=0;k<spnum[i+1];k++)
	{
		Rlocation1 = midP[(K)*7*(i+1)+k],Rlocation2 = midP[(K)*7*(i+1)+(K)*1+k],Rcolor1 = midP[(K)*7*(i+1)+(K)*2+k], Rcolor2 = midP[(K)*7*(i+1)+(K)*3+k],Rcolor3 = midP[(K)*7*(i+1)+(K)*4+k],Rcolor4 = midP[(K)*7*(i+1)+(K)*5+k],Rcolor5 = midP[(K)*7*(i+1)+(K)*6+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2)+(Lcolor4-Rcolor4)*(Lcolor4-Rcolor4)+(Lcolor5-Rcolor5)*(Lcolor5-Rcolor5));
		if (LDist<(*P1))
		{
			weight1=exp(-CDist*(*P2));
			Tweight+=weight1;
			MSS=MSS+SaA2[k]*weight1;
		}
	}
    if (i>0)
    for (k=0;k<spnum[i-1];k++)
	{
		Rlocation1 = midP[(K)*7*(i-1)+k],Rlocation2 = midP[(K)*7*(i-1)+(K)*1+k],Rcolor1 = midP[(K)*7*(i-1)+(K)*2+k], Rcolor2 = midP[(K)*7*(i-1)+(K)*3+k],Rcolor3 = midP[(K)*7*(i-1)+(K)*4+k],Rcolor4 = midP[(K)*7*(i-1)+(K)*5+k],Rcolor5 = midP[(K)*7*(i-1)+(K)*6+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2)+(Lcolor4-Rcolor4)*(Lcolor4-Rcolor4)+(Lcolor5-Rcolor5)*(Lcolor5-Rcolor5));
		if (LDist<(*P1))
		{
			weight1=exp(-CDist*(*P2));
			Tweight+=weight1;
			MSS=MSS+SaA0[k]*weight1;
		}
	}
    Result[j]=MSS/Tweight;
	return;

}
extern void colorsm2(double *Result, double* midP,double* Ind,double *K1,double *SaA,double *SaA0,double *SaA2,double *N1,double *Par1,double *Par2,double *spnum);
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
	double *midP;
    double *SaA,*SaA0,*SaA2;
	double *Result;
    double *K1,*Ind,*N1;
    double * Par1,*Par2,*spnum;
	midP=mxGetPr(prhs[0]);
    Ind=mxGetPr(prhs[1]);
    K1=mxGetPr(prhs[2]);
    SaA=mxGetPr(prhs[3]);
    SaA0=mxGetPr(prhs[4]);
    SaA2=mxGetPr(prhs[5]);
    N1=mxGetPr(prhs[6]);
    Par1=mxGetPr(prhs[7]);
    Par2=mxGetPr(prhs[8]);
    spnum=mxGetPr(prhs[9]);
    int K=(int)(*K1);
	plhs[0]=mxCreateDoubleMatrix(K,1,mxREAL);
	Result=mxGetPr(plhs[0]);
	colorsm2(Result,midP,Ind,K1,SaA,SaA0,SaA2,N1,Par1,Par2,spnum);
}
void colorsm2(double *Result, double* midP,double* Ind,double *K1,double *SaA,double *SaA0,double *SaA2,double *N1,double *Par1,double *Par2,double *spnum)
{
	double * dev_Result;
	double *dev_mid;
    double *dev_SaA,*dev_SaA0,*dev_SaA2;
    double *dev_K1,*dev_Ind,*dev_N1;
    double *dev_P1,*dev_P2,*dev_spnum;
    int K=(int)(*K1);
    int N=(int)(*N1);

	hipMalloc((void **)&dev_mid, sizeof(double)* (K) * 7 * N);
	hipMalloc((void **)&dev_SaA, sizeof(double)* K );
	hipMalloc((void **)&dev_SaA0, sizeof(double)* K );
	hipMalloc((void **)&dev_SaA2, sizeof(double)* K );
    hipMalloc((void **)&dev_K1, sizeof(double));
    hipMalloc((void **)&dev_N1, sizeof(double));
    hipMalloc((void **)&dev_Ind, sizeof(double));
	hipMalloc((void **)&dev_Result, sizeof(double)* K );
    hipMalloc((void **)&dev_P1, sizeof(double));
    hipMalloc((void **)&dev_P2, sizeof(double));
    hipMalloc((void **)&dev_spnum, sizeof(double)*N);

	hipMemcpy(dev_K1, K1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_N1, N1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_Ind, Ind, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_mid, midP, sizeof(double)* (K)*7*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_SaA, SaA, sizeof(double)* K, hipMemcpyHostToDevice);
	hipMemcpy(dev_SaA0, SaA0, sizeof(double)* K, hipMemcpyHostToDevice);
	hipMemcpy(dev_SaA2, SaA2, sizeof(double)* K, hipMemcpyHostToDevice);
	hipMemcpy(dev_P1, Par1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_P2, Par2, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_spnum, spnum, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 threads(K);
	smooth_kernel << <1, threads >> >(dev_Result,dev_mid,dev_Ind,dev_K1,dev_SaA,dev_SaA0,dev_SaA2,dev_N1,dev_P1,dev_P2,dev_spnum);

    hipMemcpy(Result, dev_Result, sizeof(double)*K, hipMemcpyDeviceToHost);

	hipFree(dev_mid);
    hipFree(dev_K1);
    hipFree(dev_SaA);
    hipFree(dev_Ind);
    hipFree(dev_Result);
    hipFree(dev_P1);
    hipFree(dev_P2);
    hipFree(dev_SaA0);
    hipFree(dev_SaA2);
    hipFree(dev_N1);
    hipFree(dev_spnum);

}	




